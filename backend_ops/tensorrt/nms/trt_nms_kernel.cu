#include "hip/hip_runtime.h"
#include <float.h>
#include <stdio.h>
#include <thrust/execution_policy.h>
#include <thrust/gather.h>
#include <thrust/sort.h>
#include <thrust/transform.h>

#include <chrono>
#include <thread>
#include <vector>

#include "common_cuda_helper.hpp"
#include "trt_nms_kernel.hpp"
#include "trt_plugin_helper.hpp"

struct NMSBox {
  float box[4];
};

struct nms_centerwh2xyxy {
  __host__ __device__ NMSBox operator()(const NMSBox box) {
    NMSBox out;
    out.box[0] = box.box[0] - box.box[2] / 2.0f;
    out.box[1] = box.box[1] - box.box[3] / 2.0f;
    out.box[2] = box.box[0] + box.box[2] / 2.0f;
    out.box[3] = box.box[1] + box.box[3] / 2.0f;
    return out;
  }
};

struct nms_sbox_idle {
  const float* idle_box_;
  __host__ __device__ nms_sbox_idle(const float* idle_box) {
    idle_box_ = idle_box;
  }

  __host__ __device__ NMSBox operator()(const NMSBox box) {
    return {idle_box_[0], idle_box_[1], idle_box_[2], idle_box_[3]};
  }
};

struct nms_score_threshold {
  float score_threshold_;
  __host__ __device__ nms_score_threshold(const float score_threshold) {
    score_threshold_ = score_threshold;
  }

  __host__ __device__ bool operator()(const float score) {
    return score < score_threshold_;
  }
};

static int const threadsPerBlock = sizeof(unsigned long long int) * 8;

__device__ inline bool devIoU(float const* const a, float const* const b,
                              const int offset, const float threshold) {
  float left = fmaxf(a[0], b[0]), right = fminf(a[2], b[2]);
  float top = fmaxf(a[1], b[1]), bottom = fminf(a[3], b[3]);
  float width = fmaxf(right - left + offset, 0.f),
        height = fmaxf(bottom - top + offset, 0.f);
  float interS = width * height;
  float Sa = (a[2] - a[0] + offset) * (a[3] - a[1] + offset);
  float Sb = (b[2] - b[0] + offset) * (b[3] - b[1] + offset);
  return interS > threshold * (Sa + Sb - interS);
}

__global__ void nms_cuda(const int n_boxes, const float iou_threshold,
                         const int offset, const float* dev_boxes,
                         unsigned long long* dev_mask) {
  const int row_start = blockIdx.y;
  const int col_start = blockIdx.x;
  const int tid = threadIdx.x;

  if (row_start > col_start) return;

  const int row_size =
      fminf(n_boxes - row_start * threadsPerBlock, threadsPerBlock);
  const int col_size =
      fminf(n_boxes - col_start * threadsPerBlock, threadsPerBlock);

  __shared__ float block_boxes[threadsPerBlock * 4];
  if (tid < col_size) {
    block_boxes[tid * 4 + 0] =
        dev_boxes[(threadsPerBlock * col_start + tid) * 4 + 0];
    block_boxes[tid * 4 + 1] =
        dev_boxes[(threadsPerBlock * col_start + tid) * 4 + 1];
    block_boxes[tid * 4 + 2] =
        dev_boxes[(threadsPerBlock * col_start + tid) * 4 + 2];
    block_boxes[tid * 4 + 3] =
        dev_boxes[(threadsPerBlock * col_start + tid) * 4 + 3];
  }
  __syncthreads();

  if (tid < row_size) {
    const int cur_box_idx = threadsPerBlock * row_start + tid;
    const float* cur_box = dev_boxes + cur_box_idx * 4;
    int i = 0;
    unsigned long long int t = 0;
    int start = 0;
    if (row_start == col_start) {
      start = tid + 1;
    }
    for (i = start; i < col_size; i++) {
      if (devIoU(cur_box, block_boxes + i * 4, offset, iou_threshold)) {
        t |= 1ULL << i;
      }
    }
    dev_mask[cur_box_idx * gridDim.y + col_start] = t;
  }
}

__global__ void nms_reindex_kernel(int n, int* output, int* index_cache) {
  CUDA_1D_KERNEL_LOOP(index, n) {
    const int old_index = output[index * 3 + 2];
    output[index * 3 + 2] = index_cache[old_index];
  }
}

__global__ void mask_to_output_kernel(const unsigned long long* dev_mask,
                                      const int* index, int* output,
                                      int* output_count, int batch_id,
                                      int cls_id, int spatial_dimension,
                                      int col_blocks,
                                      int max_output_boxes_per_class) {
  extern __shared__ unsigned long long remv[];

  // fill remv with 0
  CUDA_1D_KERNEL_LOOP(i, col_blocks) { remv[i] = 0; }
  __syncthreads();

  int start = *output_count;
  int out_per_class_count = 0;
  for (int i = 0; i < spatial_dimension; i++) {
    const int nblock = i / THREADS_PER_BLOCK;
    const int inblock = i % THREADS_PER_BLOCK;
    if (!(remv[nblock] & (1ULL << inblock))) {
      if (threadIdx.x == 0) {
        output[start * 3 + 0] = batch_id;
        output[start * 3 + 1] = cls_id;
        output[start * 3 + 2] = index[i];
        start += 1;
      }
      out_per_class_count += 1;
      if (out_per_class_count >= max_output_boxes_per_class) {
        break;
      }
      __syncthreads();
      // set every overlap box with bit 1 in remv
      const unsigned long long* p = dev_mask + i * col_blocks;
      CUDA_1D_KERNEL_LOOP(j, col_blocks) {
        if (j >= nblock) {
          remv[j] |= p[j];
        }
      }  // j
      __syncthreads();
    }
  }  // i
  if (threadIdx.x == 0) {
    *output_count = start;
  }
}

size_t get_onnxnms_workspace_size(size_t num_batches, size_t spatial_dimension,
                                  size_t num_classes, size_t boxes_word_size,
                                  int center_point_box, size_t output_length) {
  using mmlab::getAlignedSize;
  size_t boxes_xyxy_workspace = 0;
  if (center_point_box == 1) {
    boxes_xyxy_workspace =
        getAlignedSize(num_batches * spatial_dimension * 4 * boxes_word_size);
  }
  size_t scores_workspace = getAlignedSize(spatial_dimension * boxes_word_size);
  size_t boxes_workspace =
      getAlignedSize(spatial_dimension * 4 * boxes_word_size);
  const int col_blocks = DIVUP(spatial_dimension, THREADS_PER_BLOCK);
  size_t mask_workspace = getAlignedSize(spatial_dimension * col_blocks *
                                         sizeof(unsigned long long));
  size_t index_template_workspace =
      getAlignedSize(spatial_dimension * sizeof(int));
  size_t index_workspace = getAlignedSize(spatial_dimension * sizeof(int));
  size_t count_workspace = getAlignedSize(sizeof(int));
  return scores_workspace + boxes_xyxy_workspace + boxes_workspace +
         mask_workspace + index_template_workspace + index_workspace +
         count_workspace;
}

/**
 * Launch the NonMaxSuppression kernel
 *
 * The NMS will be performed on each batch/class, share the kernel implement
 * `nms_cuda`. For each batch/class, the `boxes_sorted` and `index_cache` will
 * be sorted by scores, boxes_sorted will be used in `nms_cuda` kernel. After
 * that, the output would be generated by `mask_to_output_kernel` with
 * `dev_mask` and `sorted_cache`.
 *
 * @param[in] bboxes with shape [num_batch, spatial_dimension, 4], input boxes
 * @param[in] scores with shape [num_batch, num_classes, spatial_dimension],
 *     input scores
 * @param[in] max_output_boxes_per_class max output boxes per class
 * @param[in] iou_threshold threshold of iou
 * @param[in] score_threshold threshold of scores
 * @param[in] offset box offset, only 0 or 1 is valid
 * @param[out] output with shape [output_length, 3], each row contain index
 *     (batch_id, class_id, boxes_id), filling -1 if result is not vaild.
 * @param[in] center_point_box 0 if boxes is [left, top, right, bottom] 1 if
 *     boxes is [center_x, center_y, width, height]
 * @param[in] num_batches batch size of boxes and scores
 * @param[in] spatial_dimension boxes numbers each batch
 * @param[in] num_classes class numbers
 * @param[in] output_length the max output rows
 * @param[in] workspace memory for all temporary variables.
 * @param[in] stream cuda stream
 */
void NMSCUDAKernelLauncher_float(const float* boxes, const float* scores,
                                 const int max_output_boxes_per_class,
                                 const float iou_threshold,
                                 const float score_threshold, const int offset,
                                 int* output, int center_point_box,
                                 int num_batches, int spatial_dimension,
                                 int num_classes, size_t output_length,
                                 void* workspace, hipStream_t stream) {
  using mmlab::getAlignedSize;
  const int col_blocks = DIVUP(spatial_dimension, THREADS_PER_BLOCK);
  float* boxes_sorted = (float*)workspace;
  workspace = static_cast<char*>(workspace) +
              getAlignedSize(spatial_dimension * 4 * sizeof(float));

  float* boxes_xyxy = nullptr;
  if (center_point_box == 1) {
    boxes_xyxy = (float*)workspace;
    workspace =
        static_cast<char*>(workspace) +
        getAlignedSize(num_batches * spatial_dimension * 4 * sizeof(float));
    thrust::transform(thrust::cuda::par.on(stream), (NMSBox*)boxes,
                      (NMSBox*)(boxes + num_batches * spatial_dimension * 4),
                      (NMSBox*)boxes_xyxy, nms_centerwh2xyxy());
    cudaCheckError();
  }

  float* scores_sorted = (float*)workspace;
  workspace = static_cast<char*>(workspace) +
              getAlignedSize(spatial_dimension * sizeof(float));

  unsigned long long* dev_mask = (unsigned long long*)workspace;
  workspace = static_cast<char*>(workspace) +
              getAlignedSize(spatial_dimension * col_blocks *
                             sizeof(unsigned long long));

  int* index_cache = (int*)workspace;
  workspace = static_cast<char*>(workspace) +
              getAlignedSize(spatial_dimension * sizeof(int));

  // generate sequence [0,1,2,3,4 ....]
  int* index_template = (int*)workspace;
  workspace = static_cast<char*>(workspace) +
              getAlignedSize(spatial_dimension * sizeof(int));
  thrust::sequence(thrust::cuda::par.on(stream), index_template,
                   index_template + spatial_dimension, 0);

  int max_output_boxes_per_class_cpu = max_output_boxes_per_class;
  if (max_output_boxes_per_class_cpu <= 0) {
    max_output_boxes_per_class_cpu = spatial_dimension;
  }

  int* output_count = (int*)workspace;
  workspace = static_cast<char*>(workspace) + getAlignedSize(sizeof(int));
  hipMemsetAsync(output_count, 0, sizeof(int), stream);

  // fill output with -1
  thrust::fill(thrust::cuda::par.on(stream), output, output + output_length * 3,
               -1);
  cudaCheckError();

  dim3 blocks(col_blocks, col_blocks);
  dim3 threads(THREADS_PER_BLOCK);

  for (int batch_id = 0; batch_id < num_batches; ++batch_id) {
    for (int cls_id = 0; cls_id < num_classes; ++cls_id) {
      const int batch_cls_id = batch_id * num_classes + cls_id;

      // sort boxes by score
      hipMemcpyAsync(scores_sorted, scores + batch_cls_id * spatial_dimension,
                      spatial_dimension * sizeof(float),
                      hipMemcpyDeviceToDevice, stream);
      cudaCheckError();

      hipMemcpyAsync(index_cache, index_template,
                      spatial_dimension * sizeof(int), hipMemcpyDeviceToDevice,
                      stream);
      cudaCheckError();

      thrust::sort_by_key(thrust::cuda::par.on(stream), scores_sorted,
                          scores_sorted + spatial_dimension, index_cache,
                          thrust::greater<float>());

      if (center_point_box == 1) {
        thrust::gather(thrust::cuda::par.on(stream), index_cache,
                       index_cache + spatial_dimension,
                       (NMSBox*)(boxes_xyxy + batch_id * spatial_dimension * 4),
                       (NMSBox*)boxes_sorted);
      } else {
        thrust::gather(thrust::cuda::par.on(stream), index_cache,
                       index_cache + spatial_dimension,
                       (NMSBox*)(boxes + batch_id * spatial_dimension * 4),
                       (NMSBox*)boxes_sorted);
      }

      cudaCheckError();

      if (score_threshold > 0.0f) {
        thrust::transform_if(
            thrust::cuda::par.on(stream), (NMSBox*)boxes_sorted,
            (NMSBox*)(boxes_sorted + spatial_dimension * 4), scores_sorted,
            (NMSBox*)boxes_sorted, nms_sbox_idle(boxes_sorted),
            nms_score_threshold(score_threshold));
      }

      nms_cuda<<<blocks, threads, 0, stream>>>(spatial_dimension, iou_threshold,
                                               offset, boxes_sorted, dev_mask);

      // will be performed when dev_mask is full.
      mask_to_output_kernel<<<1, THREADS_PER_BLOCK,
                              col_blocks * sizeof(unsigned long long),
                              stream>>>(
          dev_mask, index_cache, output, output_count, batch_id, cls_id,
          spatial_dimension, col_blocks, max_output_boxes_per_class_cpu);
    }  // cls_id
  }    // batch_id
}
